//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hipsolver.h>
//#include <device_functions.h>

//constants
#define WIDTH 32
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 40

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

//idx has to allocate mxn values
//d has to allocate mxn values
__global__
void knn(float* Dt, int n, float* M, int m, int* idx, int k, float* d)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j, s;
	float key = 0.0f;

	for (j = 0; j < m; j++)
		d[j + i * m] = (float)sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));

	__syncthreads();

	//sort the distances saving the index values (insertion sort)
	//each thread is in charge of a distance sort
	float* arr = d + i * m;
	int* r = idx + i * m;
	r[0] = 0;
	for (s = 0; s < m; s++)
	{
		key = arr[s];
		j = s - 1;
		while (j >= 0 && arr[j] > key)
		{
			arr[j + 1] = arr[j];
			r[j + 1] = r[j];
			j--;
		}
		arr[j + 1] = key;
		r[j + 1] = s;
	}
}

__global__
void Normals(float* q , int* neighbors, int n, int m, int k, float* bar, float* A_total, float* normals)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = 0, stride = 0;

	//printf("%d\n", i);
	//step 1: find the centroid of the k nearest neighbors
	for (j = 1; j < k + 1; j++)
	{
		stride = neighbors[j + i * m];//neighbors are stored row-major
		bar[0 + 3 * i] += (q[0 + stride * 3] / (float)k);//q is stored colum-major (x1y1z1 ...)
		bar[1 + 3 * i] += (q[1 + stride * 3] / (float)k);
		bar[2 + 3 * i] += (q[2 + stride * 3] / (float)k);
	}
	//for (j = 0; j < 3; j++) printf("bar%d[%d]: %.3f\n", j, i, bar[j + i * 3]);
	__syncthreads();

	//step 2: find the covariance matrix A
	for (j = 1; j < k + 1; j++)
	{
		stride = neighbors[j + i * m];
		//place the values of the upper triangular matrix A only
		A_total[0 + 9 * i] += (q[0 + stride * 3] - bar[0]) * (q[0 + stride * 3] - bar[0]);
		A_total[1 + 9 * i] += (q[0 + stride * 3] - bar[0]) * (q[1 + stride * 3] - bar[1]);
		A_total[2 + 9 * i] += (q[0 + stride * 3] - bar[0]) * (q[2 + stride * 3] - bar[2]);
		A_total[4 + 9 * i] += (q[1 + stride * 3] - bar[1]) * (q[1 + stride * 3] - bar[1]);
		A_total[5 + 9 * i] += (q[1 + stride * 3] - bar[1]) * (q[2 + stride * 3] - bar[2]);
		A_total[8 + 9 * i] += (q[2 + stride * 3] - bar[2]) * (q[2 + stride * 3] - bar[2]);
	}
	__syncthreads();

	float* A = A_total + i * 9;
	//step 3: compute the eigenvectors of A
	float p1 = A[1] * A[1] + A[2] * A[2] + A[5] * A[5];
	float qi = 0.0f, p2 = 0.0f, p = 0.0f, r = 0.0f, phi = 0.0f;
	float eigen[3] = {};

	qi = (A[0] + A[4] + A[8]) / 3.0f;//trace(A)
	p2 = (A[0] - qi) * (A[0] - qi) +
		(A[4] - qi) * (A[4] - qi) +
		(A[8] - qi) * (A[8] - qi) + 2 * p1;
	p = (float)sqrt(p2 / 6.0f);
	r = ((float)1 / (2 * p * p * p)) *
		((A[0] - qi) * ((A[4] - qi) * (A[8] - qi) - A[5] * A[5])
			- A[1] * (A[1] * (A[8] - qi) - A[2] * A[5])
			+ A[2] * (A[1] * A[5] - A[2] * (A[4] - qi)));
	if (r <= -1) phi = (float)M_PI / 3.0f;
	else if (r >= 1) phi = 0.0f;
	else  phi = (float)acos(r) / 3.0f;

	//the eigenvalues satisfy eig3 <= eig2 <= eig1
	//eigen[0] = qi + 2 * p * (float)cos(phi);//eigenvalue 1
	eigen[2] = qi + 2 * p * (float)cos(phi + (2 * M_PI / 3));//eigenvalue 3
	//eigen[1] = 3 * qi - eigen[0] - eigen[2];//eigenvalue 2

	A[3] = A[1];
	A[0] -= eigen[2];
	A[4] -= eigen[2];
	float aux = A[3] / A[0];
	A[3] -= A[0] * aux;
	A[4] -= A[1] * aux;
	A[5] -= A[2] * aux;

	float eigenvector[3] = { 1.0f,1.0f,1.0f };
	eigenvector[1] = -A[5] / A[4];
	eigenvector[0] = -(A[1] * eigenvector[1] + A[2] * eigenvector[2]) / A[0];

	float modulo = sqrt(eigenvector[0] * eigenvector[0] + eigenvector[1] * eigenvector[1] + eigenvector[2] * eigenvector[2]);
	normals[0 + i * 3] = eigenvector[0]/ modulo;
	normals[1 + i * 3] = eigenvector[1]/ modulo;
	normals[2 + i * 3] = eigenvector[2] / modulo;
}

__global__
void Matching(float* Dt, float* M, int m, int* idx)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float min = 100000;
	float d;
	for (int j = 0; j < m; j++)
	{
		d = (float)sqrt(pow((Dt[0 + i * 3] - M[0 + j * 3]), 2) + pow((Dt[1 + i * 3] - M[1 + j * 3]), 2) + pow((Dt[2 + i * 3] - M[2 + j * 3]), 2));
		if (d < min)
		{
			min = d;
			idx[i] = j;
		}
	}
}

//C has to be stored in column-major order
__global__
void Cxb(float* p, int n, float* q, int m, int* idx, float* normals, float* cn, float* C_total, float* b_total, float* C, float* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = idx[i];
	float* aux = (float*)malloc((size_t)n * sizeof(float));
	cn[0 + i * 6] = p[1 + i * 3] * normals[2 + stride * 3] -
		p[2 + i * 3] * normals[1 + stride * 3];//cix
	cn[1 + i * 6] = p[2 + i * 3] * normals[0 + stride * 3] -
		p[0 + i * 3] * normals[2 + stride * 3];//ciy
	cn[2 + i * 6] = p[0 + i * 3] * normals[1 + stride * 3] -
		p[1 + i * 3] * normals[0 + stride * 3];//ciz
	cn[3 + i * 6] = normals[0 + stride * 3];//nix
	cn[4 + i * 6] = normals[1 + stride * 3];//niy
	cn[5 + i * 6] = normals[2 + stride * 3];//niz

	C_total[0 + i * 21] = cn[0 + i * 6] * cn[0 + i * 6]; C_total[1 + i * 21] = cn[0 + i * 6] * cn[1 + i * 6]; C_total[2 + i * 21] = cn[0 + i * 6] * cn[2 + i * 6];
	C_total[3 + i * 21] = cn[0 + i * 6] * cn[3 + i * 6]; C_total[4 + i * 21] = cn[0 + i * 6] * cn[4 + i * 6]; C_total[5 + i * 21] = cn[0 + i * 6] * cn[5 + i * 6];
	C_total[6 + i * 21] = cn[1 + i * 6] * cn[1 + i * 6]; C_total[7 + i * 21] = cn[1 + i * 6] * cn[2 + i * 6]; C_total[8 + i * 21] = cn[1 + i * 6] * cn[3 + i * 6];
	C_total[9 + i * 21] = cn[1 + i * 6] * cn[4 + i * 6]; C_total[10 + i * 21] = cn[1 + i * 6] * cn[5 + i * 6]; C_total[11 + i * 21] = cn[2 + i * 6] * cn[2 + i * 6];
	C_total[12 + i * 21] = cn[2 + i * 6] * cn[3 + i * 6]; C_total[13 + i * 21] = cn[2 + i * 6] * cn[4 + i * 6]; C_total[14 + i * 21] = cn[2 + i * 6] * cn[5 + i * 6];
	C_total[15 + i * 21] = cn[3 + i * 6] * cn[3 + i * 6]; C_total[16 + i * 21] = cn[3 + i * 6] * cn[4 + i * 6]; C_total[17 + i * 21] = cn[3 + i * 6] * cn[5 + i * 6];
	C_total[18 + i * 21] = cn[4 + i * 6] * cn[4 + i * 6]; C_total[19 + i * 21] = cn[4 + i * 6] * cn[5 + i * 6]; C_total[20 + i * 21] = cn[5 + i * 6] * cn[5 + i * 6];

	aux[i] = (p[0 + i * 3] - q[0 + i * 3]) * cn[3 + i * 6] +
		(p[1 + i * 3] - q[1 + i * 3]) * cn[4 + i * 6] +
		(p[2 + i * 3] - q[2 + i * 3]) * cn[5 + i * 6];

	b_total[0 + i * 6] = cn[0 + i * 6] * aux[i]; b_total[1 + i * 6] = cn[1 + i * 6] * aux[i]; b_total[2 + i * 6] = cn[2 + i * 6] * aux[i];
	b_total[3 + i * 6] = cn[3 + i * 6] * aux[i]; b_total[4 + i * 6] = cn[4 + i * 6] * aux[i]; b_total[5 + i * 6] = cn[5 + i * 6] * aux[i];
	__syncthreads();

	for (int s = 1; s < n; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			//C
			C_total[0 + i * 21] += C_total[0 + (i + s) * 21]; C_total[1 + i * 21] += C_total[1 + (i + s) * 21]; C_total[2 + i * 21] += C_total[2 + (i + s) * 21];
			C_total[3 + i * 21] += C_total[3 + (i + s) * 21]; C_total[4 + i * 21] += C_total[4 + (i + s) * 21]; C_total[5 + i * 21] += C_total[5 + (i + s) * 21];
			C_total[6 + i * 21] += C_total[6 + (i + s) * 21]; C_total[7 + i * 21] += C_total[7 + (i + s) * 21]; C_total[8 + i * 21] += C_total[8 + (i + s) * 21];
			C_total[9 + i * 21] += C_total[9 + (i + s) * 21]; C_total[10 + i * 21] += C_total[10 + (i + s) * 21]; C_total[11 + i * 21] += C_total[11 + (i + s) * 21];
			C_total[12 + i * 21] += C_total[12 + (i + s) * 21]; C_total[13 + i * 21] += C_total[13 + (i + s) * 21]; C_total[14 + i * 21] += C_total[14 + (i + s) * 21];
			C_total[15 + i * 21] += C_total[15 + (i + s) * 21]; C_total[16 + i * 21] += C_total[16 + (i + s) * 21]; C_total[17 + i * 21] += C_total[17 + (i + s) * 21];
			C_total[18 + i * 21] += C_total[18 + (i + s) * 21]; C_total[19 + i * 21] += C_total[19 + (i + s) * 21]; C_total[20 + i * 21] += C_total[20 + (i + s) * 21];

			//b
			b_total[0 + i * 6] += b_total[0 + (i + s) * 6]; b_total[1 + i * 6] += b_total[1 + (i + s) * 6]; b_total[2 + i * 6] += b_total[2 + (i + s) * 6];
			b_total[3 + i * 6] += b_total[3 + (i + s) * 6]; b_total[4 + i * 6] += b_total[4 + (i + s) * 6]; b_total[5 + i * 6] += b_total[5 + (i + s) * 6];
		}
		__syncthreads();
	}

	if (i == 0)
	{
		//C
		C[0] = C_total[0 + i * 21]; C[6] = C_total[1 + i * 21]; C[12] = C_total[2 + i * 21]; C[18] = C_total[3 + i * 21]; C[24] = C_total[4 + i * 21]; C[30] = C_total[5 + i * 21];
		C[7] = C_total[6 + i * 21]; C[13] = C_total[7 + i * 21]; C[19] = C_total[8 + i * 21]; C[25] = C_total[9 + i * 21]; C[31] = C_total[10 + i * 21];
		C[14] = C_total[11 + i * 21]; C[20] = C_total[12 + i * 21]; C[26] = C_total[13 + i * 21]; C[32] = C_total[14 + i * 21];
		C[21] = C_total[15 + i * 21]; C[27] = C_total[16 + i * 21]; C[33] = C_total[17 + i * 21];
		C[28] = C_total[18 + i * 21]; C[34] = C_total[19 + i * 21];
		C[35] = C_total[20 + i * 21];

		//b
		b[0] = b_total[0 + i * 6]; b[1] = b_total[1 + i * 6]; b[2] = b_total[2 + i * 6];
		b[3] = b_total[3 + i * 6]; b[4] = b_total[4 + i * 6]; b[5] = b_total[5 + i * 6];
		b[6] = b_total[6 + i * 6]; b[7] = b_total[7 + i * 6]; b[6] = b_total[0 + i * 6];
	}
	free(aux);
}

__global__
void RyT(float* R, float* T, float* P, float* Q)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	Q[0 + i * 3] = R[0 + 0 * 3] * P[0 + i * 3] + R[0 + 1 * 3] * P[1 + i * 3] + R[0 + 2 * 3] * P[2 + i * 3] + T[0];
	Q[1 + i * 3] = R[1 + 0 * 3] * P[0 + i * 3] + R[1 + 1 * 3] * P[1 + i * 3] + R[1 + 2 * 3] * P[2 + i * 3] + T[1];
	Q[2 + i * 3] = R[2 + 0 * 3] * P[0 + i * 3] + R[2 + 1 * 3] * P[1 + i * 3] + R[2 + 2 * 3] * P[2 + i * 3] + T[2];
}

__global__
void Error(int n, float* aux, float* D, float* M, int* idx, float* error, int iteration)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	aux[0 + i * 3] = pow(M[0 + idx[i] * 3] - D[0 + i * 3], 2);
	aux[1 + i * 3] = pow(M[1 + idx[i] * 3] - D[1 + i * 3], 2);
	aux[2 + i * 3] = pow(M[2 + idx[i] * 3] - D[2 + i * 3], 2);
	__syncthreads();

	for (int s = 1; s < n; s *= 2)//parallel reduction
	{
		if (i % (2 * s) == 0)
		{
			aux[0 + i * 3] += aux[0 + (i + s) * 3];
			aux[1 + i * 3] += aux[1 + (i + s) * 3];
			aux[2 + i * 3] += aux[2 + (i + s) * 3];
		}
		__syncthreads();
	}

	if (i == 0)
	{
		error[iteration] = (float)sqrt((aux[0] + aux[1] + aux[2]) / (float)n);
		//printf("Error: %f\n",error[iteration]);
	}
}

int main(void)
{
	int num_points, i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	//for this specific case the number of points of the 2 clouds are the same
	int d_points = num_points;
	int m_points = num_points;

	////////////////1st:Creation of the synthetic data//////////////

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + ((float)i * (float)lenght) / (float(n) - 1.0f);
	}

	//create the meshgrid
	float* mesh_x = (float*)malloc(num_points * sizeof(float));
	float* mesh_y = (float*)malloc(num_points * sizeof(float));

	if ((mesh_x != NULL) && (mesh_y != NULL))
	{
		i = 0;
		k = 0;
		while (i < num_points)
		{
			j = 0;
			while (j < WIDTH)
			{
				mesh_y[i] = lin_space[j];
				mesh_x[i] = lin_space[k];
				i++; j++;
			}
			k++;
		}
	}
	else return 0;

	//Create the function z = f(x,y) = x^2-y^2
	float* z = (float*)malloc(num_points * sizeof(float));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix
	size_t bytesD = (size_t)d_points * (size_t)3 * sizeof(float);
	float* h_D = (float*)malloc(bytesD);

	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	//printf("Data point cloud\n");
	//printScloud(h_D, num_points, num_points);

	//Translation values
	ti[0] = 0.8f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 0.1f;//axis x
	ri[1] = -0.1f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r,3,3);

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	size_t bytesM = (size_t)m_points * (size_t)3 * sizeof(float);
	float* h_M = (float*)malloc(bytesM);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}
	//printf("\nModel point cloud\n");
	//printScloud(h_M, m_points, m_points);

	/////////End of 1st/////////

	//since this lines 
	//p assumes the value of D
	//q assumes the value of M
	//number of p and q points
	int p_points = d_points;
	int q_points = m_points;
	float* d_p, * d_q;
	hipMalloc(&d_p, bytesD);//p points cloud
	hipMalloc(&d_q, bytesM);//p points cloud//q point cloud
	//transfer data from D and M to p and q
	hipMemcpy(d_p, h_D, bytesD, hipMemcpyHostToDevice);//copy data cloud to p
	hipMemcpy(d_q, h_M, bytesM, hipMemcpyHostToDevice);//copy model cloud to q
	hipError_t err = hipSuccess;//for checking errors in kernels

	/////////2nd: Normals estimation/////////
	int GridSize = 8;
	int BlockSize = q_points / GridSize;
	printf("For normals:\nGrid Size: %d, Block Size: %d\n", GridSize, BlockSize);

	int* d_NeighborIds = NULL;
	float* d_dist = NULL;
	size_t neighbors_size = (size_t)p_points * (size_t)q_points * sizeof(int);
	hipMalloc(&d_NeighborIds, neighbors_size);
	hipMalloc(&d_dist, (size_t)p_points * (size_t)q_points * sizeof(float));
	k = 4;//number of nearest neighbors
	float* d_bar, * d_A;
	hipMalloc(&d_bar, bytesM);
	hipMalloc(&d_A, (size_t)9 * (size_t)q_points * sizeof(float));
	float* h_A = (float*)malloc((size_t)9 * (size_t)q_points * sizeof(float));
	hipMemset(d_bar, 0, bytesM * sizeof(float));
	hipMemset(d_A, 0, (size_t)9 * (size_t)q_points * sizeof(float));

	float* d_normals = NULL;
	hipMalloc(&d_normals, bytesM);
	float* h_normals = (float*)malloc(bytesM);

	//cudaEventRecord(start);//start time normals estimation
	knn <<< GridSize, BlockSize >>> (d_q, q_points, d_q, q_points, d_NeighborIds, k + 1, d_dist);
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error in knn kernel: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	int* h_NeighborIds = (int*)malloc(neighbors_size);
	hipMemcpy(h_NeighborIds, d_NeighborIds, neighbors_size, hipMemcpyDeviceToHost);
	/*printf("Neighbor IDs:\n");
	for (i = 0; i < p_points; i++)
	{
		printf("%d: ", i + 1);
		for (j = 0; j < k + 1; j++) printf("%d ", h_NeighborIds[j + i * q_points] + 1);
		printf("\n");
	}
	printf("\n");*/
	Normals <<< GridSize, BlockSize >>> (d_q, d_NeighborIds, p_points, q_points, k, d_bar, d_A, d_normals);
	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error in normals kernel: %s\n", hipGetErrorString(err));
	hipDeviceSynchronize();
	hipMemcpy(h_A, d_A, (size_t)9 * (size_t)q_points * sizeof(float), hipMemcpyDeviceToHost);
	for (i = 0; i < q_points; i++)
	{
		printf("A[%d]:\n", i + 1);
		for (j = 0; j < 3; j++)
		{
			for (int u = 0; u < 3; u++) printf("%.4f ", h_A[j + u]);
			printf("\n");
		}
		printf("\n");
	}


	hipMemcpy(h_normals, d_normals, bytesM, hipMemcpyDeviceToHost);
	/*printf("Normals:\n");
	for (i = 0; i < q_points; i++)
	{
		printf("%d: ", i + 1);
		for (j = 0; j < 3; j++) printf("%.4f ", h_normals[j + i * 3]);
		printf("\n");
	}
	printf("\n");*/
	/////////End of 2nd/////////

	free(mesh_x), free(mesh_y), free(z);
	free(h_M), free(h_D);
	hipFree(d_p), hipFree(d_q);// cudaFree(d_aux);
	hipFree(d_NeighborIds), hipFree(d_dist);free(h_NeighborIds);
	hipFree(d_bar), hipFree(d_A);
	hipFree(d_normals); //free(h_normals);
	//cudaFree(d_idx); //free(h_idx);
	//cudaFree(d_error);
	//cudaFree(d_work), cudaFree(devInfo);
	//cudaFree(d_C), cudaFree(d_b), free(h_b);
	//cudaFree(d_cn), cudaFree(d_C_total), cudaFree(d_b_total);
	//cudaFree(d_temp_r), cudaFree(d_temp_T), free(h_temp_r), free(h_temp_T);

	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.4f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}