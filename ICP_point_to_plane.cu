//For debugging:
//nvcc ICP_standard.cu -lcublas -lcurand -lcusolver -o ICP_cuda

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hipsolver.h>
//#include <device_functions.h>

//constants
#define WIDTH 32
#define XY_min -2.0
#define XY_max 2.0
#define MAX_ITER 40

void SmatrixMul(float* A, float* B, float* C, int m, int n, int k);
void printScloud(float* cloud, int num_points, int points2show);
void printSarray(float* array, int points2show);
void printIarray(int* array, int points2show);

int main()
{
	int num_points, i, j, k;
	float ti[3], ri[3];
	float lin_space[WIDTH], lenght;

	num_points = WIDTH * WIDTH;//number of points
	lenght = XY_max - XY_min;

	//for this specific case the number of points of the 2 clouds are the same
	int d_points = num_points;
	int m_points = num_points;

	////////////////1st:Creation of the synthetic data//////////////

	//create an array with all points equally separated
	int n = WIDTH;
	for (i = 0; i < WIDTH; i++)
	{
		lin_space[i] = (float)XY_min + ((float)i * (float)lenght) / (float(n) - 1.0f);
	}

	//create the meshgrid
	float* mesh_x = (float*)malloc(num_points * sizeof(float));
	float* mesh_y = (float*)malloc(num_points * sizeof(float));

	if ((mesh_x != NULL) && (mesh_y != NULL))
	{
		i = 0;
		k = 0;
		while (i < num_points)
		{
			j = 0;
			while (j < WIDTH)
			{
				mesh_y[i] = lin_space[j];
				mesh_x[i] = lin_space[k];
				i++; j++;
			}
			k++;
		}
	}
	else return 0;

	//Create the function z = f(x,y) = x^2-y^2
	float* z = (float*)malloc(num_points * sizeof(float));
	for (i = 0; i < num_points; i++) z[i] = pow(mesh_x[i], 2) - pow(mesh_y[i], 2);

	//Create data point cloud matrix
	size_t bytesD = (size_t)d_points * (size_t)3 * sizeof(float);
	float* h_D = (float*)malloc(bytesD);
	
	k = 0;
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			if (j == 0) h_D[k] = mesh_x[i];
			if (j == 1) h_D[k] = mesh_y[i];
			if (j == 2) h_D[k] = z[i];
			k++;
		}
	}

	//printf("Data point cloud\n");
	//printScloud(h_D, num_points, num_points);

	//Translation values
	ti[0] = 1.0f;//x
	ti[1] = -0.3f;//y
	ti[2] = 0.2f;//z

	//Rotation values (rad)
	ri[0] = 1.0f;//axis x
	ri[1] = -0.5f;//axis y
	ri[2] = 0.05f;//axis z

	float h_r[9] = {};
	float cx = (float)cos(ri[0]); float cy = (float)cos(ri[1]); float cz = (float)cos(ri[2]);
	float sx = (float)sin(ri[0]); float sy = (float)sin(ri[1]); float sz = (float)sin(ri[2]);
	h_r[0] = cy * cz; h_r[1] = (cz * sx * sy) + (cx * sz); h_r[2] = -(cx * cz * sy) + (sx * sz);
	h_r[3] = -cy * sz; h_r[4] = (cx * cz) - (sx * sy * sz); h_r[5] = (cx * sy * sz) + (cz * sx);
	h_r[6] = sy; h_r[7] = -cy * sx; h_r[8] = cx * cy;
	//printf("Ri:\n");
	//printScloud(h_r,3,3);

	//Create model point cloud matrix (target point cloud)
	//every matrix is defined using the colum-major order
	size_t bytesM = (size_t)m_points * (size_t)3 * sizeof(float);
	float* h_M = (float*)malloc(bytesM);

	//h_M = h_r*h_D
	SmatrixMul(h_r, h_D, h_M, 3, num_points, 3);
	//h_M = h_M + t
	for (i = 0; i < num_points; i++)
	{
		for (j = 0; j < 3; j++)
		{
			h_M[j + i * 3] += ti[j];
		}
	}
	//printf("\nModel point cloud\n");
	//printScloud(h_M, NUM_POINTS, NUM_POINTS);

	/////////End of 1st/////////

	/////////2nd: Normals estimation/////////



	/////////End of 2nd/////////

	free(mesh_x), free(mesh_y), free(z);
	free(h_M), free(h_D);
	return 0;
}

//double matrix multiplication colum-major order
void SmatrixMul(float* A, float* B, float* C, int m, int n, int k)
{
	int i, j, q;
	float temp = 0.0f;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < m; j++)
		{
			temp = 0.0f;
			for (q = 0; q < k; q++) temp += A[j + q * m] * B[q + i * k];
			C[j + i * m] = temp;
		}
	}
}

//print matrix
void printScloud(float* cloud, int num_points, int points2show)
{
	int i, j, offset;
	printf("x\ty\tz\n");
	if (points2show <= num_points)
	{
		for (i = 0; i < points2show; i++)
		{
			for (j = 0; j < 3; j++)
			{
				offset = j + i * 3;
				printf("%.4f\t", cloud[offset]);
				if (j % 3 == 2) printf("\n");
			}
		}
	}
	else printf("The cloud can't be printed\n\n");
}

//print vector with double values
void printSarray(float* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%.4f ", array[i]);
	}
	printf("\n");
}

//print vector with integer values
void printIarray(int* array, int points2show)
{
	int i;
	for (i = 0; i < points2show; i++)
	{
		printf("%d ", array[i]);
	}
	printf("\n");
}